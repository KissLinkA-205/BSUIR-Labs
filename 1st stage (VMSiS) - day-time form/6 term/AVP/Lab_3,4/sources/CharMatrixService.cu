#include "hip/hip_runtime.h"
//
// Created by Anzhalika Dziarkach on 06.03.2022.
//

#include <iostream>
#include <chrono>
#include "CharMatrixService.cuh"

#define THREADS_PER_BLOCK  40

#define MILLISECONDS_TO_SECOND 1000

__global__ void
serpentineBypassKernelFullBlock(const char *initialMatrix, char *outputMatrix, long long lines, long long columns,
                                bool reverse) {
    long long i = blockIdx.x * blockDim.x + threadIdx.x;
    long long j = blockIdx.y * blockDim.y + threadIdx.y;
    long long initialIndex, outputIndex;

    if (reverse) {
        initialIndex = i * lines + j;
        outputIndex = (i + 1) % 2 == 0 ? i + (lines - j - 1) * columns : j * columns + i;
    } else {
        initialIndex = (i + 1) % 2 == 0 ? i + (lines - j - 1) * columns : j * columns + i;
        outputIndex = i * lines + j;
    }

    outputMatrix[outputIndex] = initialMatrix[initialIndex];
}

__global__ void
serpentineBypassKernelNotFullBlock(const char *initialMatrix, size_t initialMatrixPitch, char *outputMatrix,
                                   size_t outputMatrixPitch, long long lines, long long columns,
                                   bool reverse) {
    long long i = blockIdx.x * blockDim.x + threadIdx.x;
    long long j = blockIdx.y * blockDim.y + threadIdx.y;
    long long initialIndex, outputIndex;

    if (reverse) {
        initialIndex = j * initialMatrixPitch + i;
        outputIndex = ((j * columns + i) / lines + 1) % 2 == 0 ?
                      (lines - 1 - (j * columns + i) % lines) * outputMatrixPitch + (j * columns + i) / lines :
                      (j * columns + i) % lines * outputMatrixPitch + (j * columns + i) / lines;
    } else {
        initialIndex = ((j * columns + i) / lines + 1) % 2 == 0 ?
                       (lines - 1 - (j * columns + i) % lines) * initialMatrixPitch + (j * columns + i) / lines :
                       (j * columns + i) % lines * initialMatrixPitch + (j * columns + i) / lines;
        outputIndex = j * outputMatrixPitch + i;
    }

    if (i < columns && j < lines) {
        outputMatrix[outputIndex] = initialMatrix[initialIndex];
    }
}

__global__ void
serpentineBypassKernelSharedNotFullBlock(const char *initialMatrix, size_t initialMatrixPitch, char *outputMatrix,
                                         size_t outputMatrixPitch, long long lines, long long columns,
                                         bool reverse) {
    __shared__ char sharedMemory[THREADS_PER_BLOCK];
    long long i = blockIdx.x * blockDim.x + threadIdx.x;
    long long j = blockIdx.y * blockDim.y + threadIdx.y;
    long long initialIndex, outputIndex;

    if (reverse) {
        initialIndex = j * initialMatrixPitch + i;
        outputIndex = ((j * columns + i) / lines + 1) % 2 == 0 ?
                      (lines - 1 - (j * columns + i) % lines) * outputMatrixPitch + (j * columns + i) / lines :
                      (j * columns + i) % lines * outputMatrixPitch + (j * columns + i) / lines;
    } else {
        initialIndex = ((j * columns + i) / lines + 1) % 2 == 0 ?
                       (lines - 1 - (j * columns + i) % lines) * initialMatrixPitch + (j * columns + i) / lines :
                       (j * columns + i) % lines * initialMatrixPitch + (j * columns + i) / lines;
        outputIndex = j * outputMatrixPitch + i;
    }

    if (i < columns && j < lines) {
        sharedMemory[threadIdx.x * blockDim.y + threadIdx.y] = initialMatrix[initialIndex];
    }

    if (i < columns && j < lines) {
        outputMatrix[outputIndex] = sharedMemory[threadIdx.x * blockDim.y + threadIdx.y];
    }
}

double CharMatrixService::serpentineBypassUsingCPU(CharMatrix &initialMatrix, CharMatrix &outputMatrix, bool reverse) {
    if (initialMatrix.getColumns() != outputMatrix.getColumns() ||
        initialMatrix.getLines() != outputMatrix.getLines()) {
        std::cerr << "Exception (serpentineBypassUsingCPU): matrix sizes don't match!" << std::endl;
        return -1;
    }

    using namespace std::chrono;
    high_resolution_clock::time_point start = high_resolution_clock::now();

    long long resultPosition = 0;
    for (auto i = 0; i < initialMatrix.getColumns(); i++) {
        for (auto j = 0; j < initialMatrix.getLines(); j++) {
            if (reverse) {
                outputMatrix.getMatrix()[(i + 1) % 2 == 0 ? i + (initialMatrix.getLines() - j - 1)
                                                                * initialMatrix.getColumns() :
                                         j * initialMatrix.getColumns() +
                                         i] = initialMatrix.getMatrix()[resultPosition++];
            } else {
                outputMatrix.getMatrix()[resultPosition++] =
                        initialMatrix.getMatrix()[(i + 1) % 2 == 0 ? i + (initialMatrix.getLines() - j - 1)
                                                                         * initialMatrix.getColumns() :
                                                  j * initialMatrix.getColumns() + i];
            }
        }
    }

    high_resolution_clock::time_point end = high_resolution_clock::now();
    return duration_cast<duration<double>>(end - start).count();
}

double CharMatrixService::serpentineBypassUsingGPUFullBlock(CharMatrix &initialMatrix, CharMatrix &outputMatrix,
                                                            bool reverse) {
    if (initialMatrix.getColumns() != outputMatrix.getColumns() ||
        initialMatrix.getLines() != outputMatrix.getLines()) {
        std::cerr << "Exception (serpentineBypassUsingGPUFullBlock): matrix sizes don't match!" << std::endl;
        return -1;
    }

    hipError_t cudaStatus = hipSuccess;
    hipEvent_t start, end;
    char *initialMatrixGPU, *outputMatrixGPU;
    long long lines = initialMatrix.getLines();
    long long columns = initialMatrix.getColumns();

    hipEventCreate(&start);
    hipEventCreate(&end);

    cudaStatus = hipMalloc((void **) &initialMatrixGPU, lines * columns * sizeof(char));
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUFullBlock): hipMalloc initialMatrix failed!" << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMalloc((void **) &outputMatrixGPU, lines * columns * sizeof(char));
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUFullBlock): hipMalloc outputMatrix failed!" << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    hipEventRecord(start, nullptr);

    cudaStatus = hipMemcpy(initialMatrixGPU, initialMatrix.getMatrix(), lines * columns * sizeof(char),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUFullBlock): hipMemcpy initialMatrixGPU failed!" << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMemcpy(outputMatrixGPU, outputMatrix.getMatrix(), lines * columns * sizeof(char),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUFullBlock): hipMemcpy outputMatrixGPU failed!" << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 numBlocks(columns / threadsPerBlock.x, lines / threadsPerBlock.y);
    serpentineBypassKernelFullBlock<<<numBlocks, threadsPerBlock>>>(initialMatrixGPU,
                                                                    outputMatrixGPU, lines, columns, reverse);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUFullBlock): serpentineBypassKernelFullBlock failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMemcpy(outputMatrix.getMatrix(), outputMatrixGPU, lines * columns * sizeof(char),
                            hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUFullBlock): hipMemcpy outputMatrix failed!" << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    hipEventRecord(end, nullptr);
    hipEventSynchronize(end);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, end);

    hipFree(initialMatrixGPU);
    hipFree(outputMatrixGPU);
    return elapsedTime / MILLISECONDS_TO_SECOND;
}

double CharMatrixService::serpentineBypassUsingGPUNotFullBlock(CharMatrix &initialMatrix, CharMatrix &outputMatrix,
                                                               bool reverse) {
    if (initialMatrix.getColumns() != outputMatrix.getColumns() ||
        initialMatrix.getLines() != outputMatrix.getLines()) {
        std::cerr << "Exception (serpentineBypassUsingGPUNotFullBlock): matrix sizes don't match!" << std::endl;
        return -1;
    }

    hipError_t cudaStatus = hipSuccess;
    hipEvent_t start, end;
    char *initialMatrixGPU, *outputMatrixGPU;
    size_t initialMatrixPitch = 0, outputMatrixPitch = 0;
    long long lines = initialMatrix.getLines();
    long long columns = initialMatrix.getColumns();

    hipEventCreate(&start);
    hipEventCreate(&end);

    cudaStatus = hipMallocPitch((void **) &initialMatrixGPU, &initialMatrixPitch, columns * sizeof(char), lines);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUNotFullBlock): hipMallocPitch initialMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMallocPitch((void **) &outputMatrixGPU, &outputMatrixPitch, columns * sizeof(char), lines);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUNotFullBlock): hipMallocPitch outputMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    hipEventRecord(start, nullptr);

    cudaStatus = hipMemcpy2DAsync(initialMatrixGPU, initialMatrixPitch, initialMatrix.getMatrix(),
                                   columns * sizeof(char),
                                   columns * sizeof(char), lines, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUNotFullBlock): hipMemcpy2DAsync initialMatrixGPU failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMemcpy2DAsync(outputMatrixGPU, outputMatrixPitch, outputMatrix.getMatrix(),
                                   columns * sizeof(char),
                                   columns * sizeof(char), lines, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUNotFullBlock): hipMemcpy2DAsync outputMatrixGPU failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 numBlocks((columns + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (lines + threadsPerBlock.y - 1) / threadsPerBlock.y);
    serpentineBypassKernelNotFullBlock<<<numBlocks, threadsPerBlock>>>(initialMatrixGPU, initialMatrixPitch,
                                                                       outputMatrixGPU, outputMatrixPitch, lines,
                                                                       columns, reverse);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUNotFullBlock): serpentineBypassKernelNotFullBlock failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMemcpy2DAsync(outputMatrix.getMatrix(), columns * sizeof(char), outputMatrixGPU, outputMatrixPitch,
                                   columns * sizeof(char), lines,
                                   hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingGPUNotFullBlock): hipMemcpy2DAsync outputMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    hipEventRecord(end, nullptr);
    hipEventSynchronize(end);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, end);

    hipFree(initialMatrixGPU);
    hipFree(outputMatrixGPU);
    return elapsedTime / MILLISECONDS_TO_SECOND;
}

double CharMatrixService::serpentineBypassUsingSharedGPUNotFullBlock(CharMatrix &initialMatrix,
                                                                     CharMatrix &outputMatrix, bool reverse) {
    if (initialMatrix.getColumns() != outputMatrix.getColumns() ||
        initialMatrix.getLines() != outputMatrix.getLines()) {
        std::cerr << "Exception (serpentineBypassUsingSharedGPUNotFullBlock): matrix sizes don't match!" << std::endl;
        return -1;
    }

    hipError_t cudaStatus = hipSuccess;
    hipEvent_t start, end;
    char *initialMatrixGPU, *outputMatrixGPU;
    size_t initialMatrixPitch = 0, outputMatrixPitch = 0;
    long long lines = initialMatrix.getLines();
    long long columns = initialMatrix.getColumns();

    hipEventCreate(&start);
    hipEventCreate(&end);

    cudaStatus = hipMallocPitch((void **) &initialMatrixGPU, &initialMatrixPitch, columns * sizeof(char), lines);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingSharedGPUNotFullBlock): hipMallocPitch initialMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMallocPitch((void **) &outputMatrixGPU, &outputMatrixPitch, columns * sizeof(char), lines);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingSharedGPUNotFullBlock): hipMallocPitch outputMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    hipEventRecord(start, nullptr);

    cudaStatus = hipMemcpy2DAsync(initialMatrixGPU, initialMatrixPitch, initialMatrix.getMatrix(),
                                   columns * sizeof(char),
                                   columns * sizeof(char), lines, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr
                << "Exception (serpentineBypassUsingSharedGPUNotFullBlock): hipMemcpy2DAsync initialMatrixGPU failed!"
                << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMemcpy2DAsync(outputMatrixGPU, outputMatrixPitch, outputMatrix.getMatrix(),
                                   columns * sizeof(char),
                                   columns * sizeof(char), lines, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingSharedGPUNotFullBlock): hipMemcpy2DAsync outputMatrixGPU failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 numBlocks((columns + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (lines + threadsPerBlock.y - 1) / threadsPerBlock.y);
    serpentineBypassKernelSharedNotFullBlock<<<numBlocks, threadsPerBlock>>>(initialMatrixGPU, initialMatrixPitch,
                                                                             outputMatrixGPU, outputMatrixPitch, lines,
                                                                             columns, reverse);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr
                << "Exception (serpentineBypassUsingSharedGPUNotFullBlock): serpentineBypassKernelSharedNotFullBlock failed!"
                << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMemcpy2D(outputMatrix.getMatrix(), columns * sizeof(char), outputMatrixGPU, outputMatrixPitch,
                              columns * sizeof(char), lines, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingSharedGPUNotFullBlock): hipMemcpy2DAsync outputMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    hipEventRecord(end, nullptr);
    hipEventSynchronize(end);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, end);

    hipFree(initialMatrixGPU);
    hipFree(outputMatrixGPU);
    return elapsedTime / MILLISECONDS_TO_SECOND;
}

double CharMatrixService::serpentineBypassUsingPinnedGPUNotFullBlock(CharMatrix &initialMatrix,
                                                                     CharMatrix &outputMatrix, bool reverse) {
    if (initialMatrix.getColumns() != outputMatrix.getColumns() ||
        initialMatrix.getLines() != outputMatrix.getLines()) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): matrix sizes don't match!" << std::endl;
        return -1;
    }

    hipError_t cudaStatus = hipSuccess;
    hipEvent_t start, end;
    char *initialMatrixGPU, *outputMatrixGPU;
    size_t initialMatrixPitch = 0, outputMatrixPitch = 0;
    long long lines = initialMatrix.getLines();
    long long columns = initialMatrix.getColumns();

    hipEventCreate(&start);
    hipEventCreate(&end);

    cudaStatus = hipHostRegister(initialMatrix.getMatrix(), columns * lines * sizeof(char), 0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipHostRegister initialMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipHostRegister(outputMatrix.getMatrix(), columns * lines * sizeof(char), 0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipHostRegister outputMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMallocPitch((void **) &initialMatrixGPU, &initialMatrixPitch, columns * sizeof(char), lines);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipMallocPitch initialMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMallocPitch((void **) &outputMatrixGPU, &outputMatrixPitch, columns * sizeof(char), lines);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipMallocPitch outputMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    hipEventRecord(start, nullptr);

    cudaStatus = hipMemcpy2DAsync(initialMatrixGPU, initialMatrixPitch, initialMatrix.getMatrix(),
                                   columns * sizeof(char),
                                   columns * sizeof(char), lines, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr
                << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipMemcpy2DAsync initialMatrixGPU failed!"
                << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMemcpy2DAsync(outputMatrixGPU, outputMatrixPitch, outputMatrix.getMatrix(),
                                   columns * sizeof(char),
                                   columns * sizeof(char), lines, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipMemcpy2DAsync outputMatrixGPU failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 numBlocks((columns + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (lines + threadsPerBlock.y - 1) / threadsPerBlock.y);
    serpentineBypassKernelSharedNotFullBlock<<<numBlocks, threadsPerBlock>>>(initialMatrixGPU, initialMatrixPitch,
                                                                             outputMatrixGPU, outputMatrixPitch, lines,
                                                                             columns, reverse);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr
                << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): serpentineBypassKernelSharedNotFullBlock failed!"
                << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipMemcpy2D(outputMatrix.getMatrix(), columns * sizeof(char), outputMatrixGPU, outputMatrixPitch,
                              columns * sizeof(char), lines, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipMemcpy2DAsync outputMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipHostUnregister(initialMatrix.getMatrix());
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipHostUnregister initialMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    cudaStatus = hipHostUnregister(outputMatrix.getMatrix());
    if (cudaStatus != hipSuccess) {
        std::cerr << "Exception (serpentineBypassUsingPinnedGPUNotFullBlock): hipHostUnregister outputMatrix failed!"
                  << std::endl;
        hipFree(initialMatrixGPU);
        hipFree(outputMatrixGPU);
        return -1;
    }

    hipEventRecord(end, nullptr);
    hipEventSynchronize(end);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, end);

    hipFree(initialMatrixGPU);
    hipFree(outputMatrixGPU);
    return elapsedTime / MILLISECONDS_TO_SECOND;
}
